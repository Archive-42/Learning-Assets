#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

namespace at {
namespace native {

// Some cuBLAS and cuSOLVER batched routines require input to be a device array of pointers to device individual matrices
// 'input' must be a contiguous tensor
template <typename scalar_t>
static Tensor get_device_pointers(const Tensor& input) {
  auto input_data = input.data_ptr<scalar_t>();
  int64_t input_mat_stride = matrixStride(input);

  // cublas/cusolver interface requires 'int'
  int batch_size = cuda_int_cast(batchCount(input), "batch_size");

  // if batch_size==0, then start=0 and end=0
  // if input_mat_stride==0, then step=sizeof(scalar_t)
  return at::arange(
      /*start=*/reinterpret_cast<int64_t>(input_data),
      /*end=*/reinterpret_cast<int64_t>(input_data + batch_size * input_mat_stride),
      /*step=*/static_cast<int64_t>(std::max<int64_t>(input_mat_stride, 1) * sizeof(scalar_t)),
      input.options().dtype(at::kLong));
}

template <typename scalar_t>
static void apply_triangular_solve(Tensor& A, Tensor& B, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  trans = conjugate_transpose ? HIPBLAS_OP_C : trans;
  hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;

  auto A_data = A.data_ptr<scalar_t>();
  auto B_data = B.data_ptr<scalar_t>();
  auto A_mat_stride = matrixStride(A);
  auto B_mat_stride = matrixStride(B);
  auto batch_size = batchCount(A);
  auto n = cuda_int_cast(A.size(-2), "n");
  auto nrhs = cuda_int_cast(B.size(-1), "nrhs");
  auto lda = std::max<int>(1, n);

  auto alpha = scalar_t{1};

  for (decltype(batch_size) i = 0; i < batch_size; i++) {
    scalar_t* A_working_ptr = &A_data[i * A_mat_stride];
    scalar_t* B_working_ptr = &B_data[i * B_mat_stride];
    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::trsm(handle, side, uplo, trans, diag, n, nrhs, &alpha, A_working_ptr, lda, B_working_ptr, lda);
  }
}

void triangular_solve_cublas(Tensor& A, Tensor& B, Tensor& infos, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  (void)infos; // unused
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(A.scalar_type(), "triangular_solve_cuda", [&]{
    apply_triangular_solve<scalar_t>(A, B, upper, transpose, conjugate_transpose, unitriangular);
  });
}

template <typename scalar_t>
static void apply_triangular_solve_batched(Tensor& A, Tensor& B, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  hipblasFillMode_t uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  trans = conjugate_transpose ? HIPBLAS_OP_C : trans;
  hipblasDiagType_t diag = unitriangular ? HIPBLAS_DIAG_UNIT : HIPBLAS_DIAG_NON_UNIT;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;

  auto A_data = A.data_ptr<scalar_t>();
  auto B_data = B.data_ptr<scalar_t>();
  auto A_mat_stride = matrixStride(A);
  auto B_mat_stride = matrixStride(B);
  auto batch_size = cuda_int_cast(batchCount(A), "batch_size");
  auto n = cuda_int_cast(A.size(-2), "n");
  auto nrhs = cuda_int_cast(B.size(-1), "nrhs");
  auto lda = std::max<int>(1, n);

  auto alpha = scalar_t{1};

  // cuBLAS batched trsm requires input to be the device array of pointers to device single matrices
  Tensor A_ptr_array = get_device_pointers<scalar_t>(A);
  Tensor B_ptr_array = get_device_pointers<scalar_t>(B);
  auto A_ptr_array_data = reinterpret_cast<scalar_t**>(A_ptr_array.data_ptr());
  auto B_ptr_array_data = reinterpret_cast<scalar_t**>(B_ptr_array.data_ptr());

  auto handle = at::cuda::getCurrentCUDABlasHandle();
  at::cuda::blas::trsmBatched(handle, side, uplo, trans, diag, n, nrhs, &alpha, A_ptr_array_data, lda, B_ptr_array_data, lda, batch_size);
}

void triangular_solve_batched_cublas(Tensor& A, Tensor& B, Tensor& infos, bool upper, bool transpose, bool conjugate_transpose, bool unitriangular) {
  (void)infos; // unused
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(A.scalar_type(), "triangular_solve_cuda", [&]{
    apply_triangular_solve_batched<scalar_t>(A, B, upper, transpose, conjugate_transpose, unitriangular);
  });
}

#ifdef USE_CUSOLVER

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_getrf_ptr, int* info_getrs_ptr, int n, int lda) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, info_getrf_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, lda, ipiv_ptr, self_inv_ptr, lda, info_getrs_ptr);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  const int lda = std::max<int>(1, n);

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto infos_getrf_data = infos_getrf.data_ptr<int>();
  auto infos_getrs_data = infos_getrs.data_ptr<int>();

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  // Heuristic: For small batch size or large matrix size, we use for-loop to iterate over the batches instead of
  //            calling the batched cublas routine.
  if (batch_size <= 8 || /* batch_size > 8 && */ n >= 512) {
    for (int64_t i = 0; i < batch_size; i++) {
      auto dataPtr = allocator.allocate(sizeof(int) * lda);
      int* pivot = reinterpret_cast<int*>(dataPtr.get());

      int* infos_getrf_working_ptr = &infos_getrf_data[i];
      int* infos_getrs_working_ptr = &infos_getrs_data[i];

      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, infos_getrf_working_ptr, infos_getrs_working_ptr, n, lda);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<int64_t>(self_data),
      reinterpret_cast<int64_t>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<int64_t>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<int64_t>(self_inv_data),
      reinterpret_cast<int64_t>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<int64_t>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    auto dataPtr = allocator.allocate(sizeof(int)*batch_size*lda);
    int* ipiv_array = reinterpret_cast<int*>(dataPtr.get());

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, infos_getrf_data, batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), lda,
      ipiv_array, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()), lda, infos_getrs_data, batch_size);
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, Tensor& infos_getrf, Tensor& infos_getrs) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");
  int lda = std::max<int>(1, n);

  Tensor ipiv = at::empty({lda}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), infos_getrf.data_ptr<int>(), infos_getrs.data_ptr<int>(), n, lda);
}

// This is a type dispatching helper function for 'apply_batched_inverse_lib' and 'apply_single_inverse_lib'
Tensor& _linalg_inv_out_helper_cuda_lib(Tensor& result, Tensor& infos_getrf, Tensor& infos_getrs) {
  // assuming result is in column major order and contains the matrices to invert
  Tensor input_working_copy = cloneBatchedColumnMajor(result);

  // for getrf + getrs (cusolver path)
  // result should be filled with identity matrices
  result.zero_();
  result.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);

  const int batch_size = cuda_int_cast(batchCount(result), "batchCount");

  if (result.dim() > 2) {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        input_working_copy, result, infos_getrf, infos_getrs);
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "linalg_inv_out_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(input_working_copy, result, infos_getrf, infos_getrs);
    });
  }

  return result;
}

// entrance of calculations of `inverse` using cusolver getrf + getrs, cublas getrfBatched + getriBatched
Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos_getrf = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({std::max<int64_t>(1, batchCount(self))}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  } else {
    Tensor infos_getrf = at::zeros({1}, self.options().dtype(kInt));
    Tensor infos_getrs = at::zeros({1}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, infos_getrf, infos_getrs);
    });
    batchCheckErrors(infos_getrf, "inverse_cuda");
    batchCheckErrors(infos_getrs, "inverse_cuda");
  }

  return self_inv_working_copy;
}

// call cusolver gesvdj function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  for(int i = 0; i < batchsize; i++){
    // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
    hipsolverGesvdjInfo_t gesvdj_params;
    TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
    // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));

    auto handle = at::cuda::getCurrentCUDASolverDnHandle();
    auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
    at::cuda::solver::gesvdj<scalar_t>(
      handle, jobz, /*econ=*/ some ? 1 : 0, m, n,
      self_data + i * self_stride,
      lda,
      S_data + i * S_stride,
      U_data + i * U_stride,
      lda,
      VT_data + i * VT_stride,
      ldvt,
      infos.data_ptr<int>() + i,
      gesvdj_params
    );

    TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
  }
}

// wrapper around _apply_svd_lib_gesvdj that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdj(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv, bool some) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdj", [&] {
    _apply_svd_lib_gesvdj<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv, some);
  });
}

// call cusolver gesvdj batched function to calculate svd
template<typename scalar_t>
inline static void _apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto U_data = U.data_ptr<scalar_t>();
  auto S_data = S.data_ptr<value_t>();
  auto VT_data = VT.data_ptr<scalar_t>();
  auto self_stride = matrixStride(self);
  auto U_stride = matrixStride(U);
  auto S_stride = S.size(-1);
  auto VT_stride = matrixStride(VT);

  int batchsize = cuda_int_cast(batchCount(self), "batch size");
  int m = cuda_int_cast(self.size(-2), "m");
  int n = cuda_int_cast(self.size(-1), "n");
  int lda = std::max<int>(1, m);
  int ldvt = std::max<int>(1, n);

  TORCH_INTERNAL_ASSERT(m <= 32 && n <= 32, "gesvdjBatched requires both matrix dimensions not greater than 32, but got "
                        "m = ", m, " n = ", n);

  // gesvdj_params controls the numerical accuracy of cusolver gesvdj iterations on GPU
  hipsolverGesvdjInfo_t gesvdj_params;
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateGesvdjInfo(&gesvdj_params));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetTolerance(gesvdj_params, 1.0e-7));
  // TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetMaxSweeps(gesvdj_params, 15));
  TORCH_CUSOLVER_CHECK(hipsolverDnXgesvdjSetSortEig(gesvdj_params, 1));

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  auto jobz = compute_uv ? HIPSOLVER_EIG_MODE_VECTOR : HIPSOLVER_EIG_MODE_NOVECTOR;
  at::cuda::solver::gesvdjBatched<scalar_t>(
    handle, jobz, m, n, self_data, lda, S_data, U_data, lda, VT_data, ldvt,
    infos.data_ptr<int>(), gesvdj_params, batchsize
  );

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyGesvdjInfo(gesvdj_params));
}

// wrapper around _apply_svd_lib_gesvdjBatched that handles dtype dispatch,
// creates a working copy of the input, and creates V^H from the V returned by gesvdj
inline static void apply_svd_lib_gesvdjBatched(const Tensor& self, Tensor& U, Tensor& S, Tensor& VT, Tensor& infos, bool compute_uv) {
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  VT = VT.transpose(-2, -1);  // gesvdj returns V instead of V^H

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "svd_cuda_gesvdjBatched", [&] {
    _apply_svd_lib_gesvdjBatched<scalar_t>(self_working_copy, U, S, VT, infos, compute_uv);
  });
}

// entrance of calculations of `svd` using cusolver gesvdj and gesvdjBatched
std::tuple<Tensor, Tensor, Tensor> _svd_helper_cuda_lib(const Tensor& self, bool some, bool compute_uv) {
  const int64_t batch_size = batchCount(self);
  at::Tensor infos = at::zeros({batch_size}, self.options().dtype(at::kInt));
  const int64_t m = self.size(-2);
  const int64_t n = self.size(-1);
  const int64_t k = std::min(m, n);

  Tensor U_working_copy, S_working_copy, VT_working_copy;
  std::tie(U_working_copy, S_working_copy, VT_working_copy) = \
    _create_U_S_VT(self, some, compute_uv, /* svd_use_cusolver = */ true);
  // U, S, V working copies are already column majored now

  // heuristic for using `gesvdjBatched` over `gesvdj`
  if (m <= 32 && n <= 32 && batch_size > 1 && (!some || m == n)) {
    apply_svd_lib_gesvdjBatched(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv);
  } else {
    apply_svd_lib_gesvdj(self, U_working_copy, S_working_copy, VT_working_copy, infos, compute_uv, some);
  }

  // A device-host sync will be performed.
  batchCheckErrors(infos, "svd_cuda");

  if (!compute_uv) {
    VT_working_copy.zero_();
    U_working_copy.zero_();
  }

  if (some) {
    VT_working_copy = VT_working_copy.narrow(-2, 0, k);
  }

  // so far we have computed VT, but torch.svd returns V instead. Adjust accordingly.
  VT_working_copy.transpose_(-2, -1);
  return std::make_tuple(U_working_copy, S_working_copy, VT_working_copy);
}


// Todo: cusolverDnXpotrfBatched has some numerical issue and is not used here.
//     A loop of hipsolverDnXpotrf is used in case MAGMA is not linked in the pytorch build.
//     We will switch to cusolverDnXpotrfBatched after the issue is fixed.
//     See https://github.com/pytorch/pytorch/issues/53879.
template<typename scalar_t>
inline static void apply_cholesky_cusolver_potrf(Tensor& self_working_copy, bool upper, Tensor& infos) {
  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  const auto uplo = upper ? HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
  const int64_t n = self_working_copy.size(-1);
  const int64_t lda = std::max<int64_t>(1, n);
  const int64_t batch_size = batchCount(self_working_copy);
  const int64_t matrix_stride = matrixStride(self_working_copy);

  scalar_t* self_working_copy_ptr = self_working_copy.data_ptr<scalar_t>();
  int* infos_ptr = infos.data_ptr<int>();

#ifdef USE_CUSOLVER_64_BIT
  size_t worksize_device;
  size_t worksize_host;
  hipsolverDnParams_t params;
  hipDataType datatype = at::cuda::solver::get_cusolver_datatype<scalar_t>();
  TORCH_CUSOLVER_CHECK(hipsolverDnCreateParams(&params));
  at::cuda::solver::xpotrf_buffersize(handle, params, uplo, n, datatype, nullptr, lda, datatype, &worksize_device, &worksize_host);

  // allocate workspace storage
  auto& device_allocator = *at::cuda::getCUDADeviceAllocator();
  auto workdata_device = device_allocator.allocate(worksize_device * batch_size);
  void* workdata_device_ptr = workdata_device.get();

  auto& host_allocator = *at::getCPUAllocator();
  auto workdata_host = host_allocator.allocate(worksize_host * batch_size);
  void* workdata_host_ptr = workdata_host.get();

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::xpotrf(
      handle, params, uplo, n, datatype,
      self_working_copy_ptr + i * matrix_stride,
      lda, datatype,
      (char*)workdata_device_ptr + i * worksize_device, worksize_device,
      (char*)workdata_host_ptr + i * worksize_host, worksize_host,
      infos_ptr + i
    );
  }

  TORCH_CUSOLVER_CHECK(hipsolverDnDestroyParams(params));
#else // USE_CUSOLVER_64_BIT
  int n_32 = cuda_int_cast(n, "n");
  int lda_32 = cuda_int_cast(lda, "lda");
  int lwork;
  at::cuda::solver::potrf_buffersize<scalar_t>(
    handle, uplo, n_32, nullptr, lda_32, &lwork);

   // allocate workspace storage
  auto& allocator = *at::cuda::getCUDADeviceAllocator();
  auto work_data = allocator.allocate(sizeof(scalar_t)*lwork * batch_size);
  scalar_t* work_data_ptr = static_cast<scalar_t*>(work_data.get());

  for (int64_t i = 0; i < batch_size; i++) {
    at::cuda::solver::potrf<scalar_t>(
      handle, uplo, n_32,
      self_working_copy_ptr + i * matrix_stride,
      lda_32,
      work_data_ptr + i * lwork,
      lwork,
      infos_ptr + i
    );
  }
#endif // USE_CUSOLVER_64_BIT
}

Tensor _cholesky_helper_cuda_cusolver(const Tensor& self, bool upper) {
  const int64_t batch_size = batchCount(self);
  at::Tensor infos = at::zeros({batch_size}, self.options().dtype(at::kInt));
  at::Tensor self_working_copy = cloneBatchedColumnMajor(self);

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "cholesky_cuda_potrf", [&] {
    apply_cholesky_cusolver_potrf<scalar_t>(self_working_copy, upper, infos);
  });

  batchCheckErrors(infos, "cholesky_cuda");

  return self_working_copy;
}

/*
  The orgqr function allows reconstruction of an orthogonal (or unitary) matrix Q,
  from a sequence of elementary reflectors, such as produced by the geqrf function.

  Args:
  * `self` - Tensor with the directions of the elementary reflectors below the diagonal,
              it will be overwritten with the result
  * `tau` - Tensor containing the magnitudes of the elementary reflectors
  * `infos` - Tensor to store cuSOLVER's error codes
  * `n_columns` - The number of columns of Q to be computed

  For further details, please see the cuSOLVER documentation for ORGQR and UNGQR.
*/
template <typename scalar_t>
inline void apply_orgqr_cusolver(Tensor& self, const Tensor& tau, Tensor& infos, int64_t n_columns) {
  using value_t = typename c10::scalar_value_type<scalar_t>::type;
  auto self_data = self.data_ptr<scalar_t>();
  auto tau_data = tau.data_ptr<scalar_t>();
  auto infos_data = infos.data_ptr<int>();
  auto self_matrix_stride = matrixStride(self);
  auto batchsize = cuda_int_cast(batchCount(self), "batch size");
  auto m = cuda_int_cast(self.size(-2), "m");
  auto n = cuda_int_cast(n_columns, "n");
  auto k = cuda_int_cast(tau.size(-1), "k");
  auto tau_stride = std::max<int>(1, k);
  auto lda = std::max<int>(1, m);

  // LAPACK's requirement
  TORCH_INTERNAL_ASSERT(m >= n);
  TORCH_INTERNAL_ASSERT(n >= k);

  // cuSOLVER doesn't compute anything for this case, which is wrong
  // the result should be a matrix with 1 on the diagonal
  if (k == 0) {
    self.fill_(0);
    self.diagonal(/*offset=*/0, /*dim1=*/-2, /*dim2=*/-1).fill_(1);
    return;
  }

  // get the optimal work size and allocate workspace tensor
  int lwork;
  at::cuda::solver::orgqr_buffersize<scalar_t>(
    at::cuda::getCurrentCUDASolverDnHandle(), m, n, k, self_data, lda, tau_data, &lwork);

  for (auto i = decltype(batchsize){0}; i < batchsize; i++) {
    scalar_t* self_working_ptr = &self_data[i * self_matrix_stride];
    scalar_t* tau_working_ptr = &tau_data[i * tau_stride];
    int* info_working_ptr = &infos_data[i];
    auto handle = at::cuda::getCurrentCUDASolverDnHandle();

    // allocate workspace storage
    auto& allocator = *at::cuda::getCUDADeviceAllocator();
    auto work_data = allocator.allocate(sizeof(scalar_t)*lwork);

    at::cuda::solver::orgqr<scalar_t>(
      handle, m, n, k,
      self_working_ptr,
      lda,
      tau_working_ptr,
      static_cast<scalar_t*>(work_data.get()),
      lwork,
      info_working_ptr
    );
  }
}

// This is a type dispatching helper function for 'apply_orgqr_cusolver'
Tensor& orgqr_helper_cuda_lib(Tensor& result, const Tensor& tau, Tensor& infos, int64_t n_columns) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(result.scalar_type(), "orgqr_cuda", [&]{
    apply_orgqr_cusolver<scalar_t>(result, tau, infos, n_columns);
  });
  return result;
}

#endif  // USE_CUSOLVER

}} // namespace at::native
